
#include <hip/hip_runtime.h>
#include <algorithm>
#include <stdio.h>
#include <math.h>
#include <omp.h>
//#include "utils.h"

#define BLOCK_DIM 34


double norm(double *err, long N) {
  double sum = 0;
  #pragma omp parallel for reduction (+:sum)
  for (long i = 0; i < N*N ; i+=1) {
     sum += err[i]*err[i];
     //printf("sum = %f\n",sum);
  }
  return sqrt(sum);
} 

/*  
__global__ void jacobi_kernel(double* u, double* f, long N){
  __shared__ double smem[BLOCK_DIM][BLOCK_DIM];
 long grid_dim = (N-2)/(BLOCK_DIM - 2) ;
 int idx = (blockIdx.x)*(BLOCK_DIM-2) + threadIdx.x;
 int idy = (blockIdx.y)*(BLOCK_DIM-2) + threadIdx.y;
 smem[threadIdx.x+1][threadIdx.y+1] = u[idx][idy];
 
 if (blockIdx.x == 0) smem[0][threadIdx.y] = 0;
 if (blockIdx.y == 0) smem[threadIdx.x][0] = 0;
 if (blockIdx.x == grid_dim) smem[grid_dim][threadIdx.y] = 0;
 if (blockIdx.y == grid_dim) smem[threadIdx.x][grid_dim]= 0;

}
*/

__global__ void jacobi_kernel_nsmem(double* u, double* f, double * err, double * temp, long N, double h){
 int idx = (blockIdx.x)*(BLOCK_DIM-2) + threadIdx.x + 1;
 int idy = (blockIdx.y)*(BLOCK_DIM-2) + threadIdx.y + 1;
 temp[idx*N+idy] = (h*h*f[idx*N+idy] + u[(idx-1)*N+idy] + u[idx*N+idy-1] + u[(idx+1)*N+idy] + u[idx*N+idy+1])/4;
 //if (threadIdx.x == 0) {
   //printf("temp = %f", temp[idx*N+idy]); 
 //}
 __syncthreads();
 u[idx*N + idy] = temp[idx*N+idy];
 //printf("f, idx, idy = %f %d %d\n ",f[idx*N+idy], idx, idy);
 err[idx*N+idy] = (-u[(idx-1)*N+idy] - u[idx*N+idy-1] + 4*u[idx*N+idy] - u[(idx+1)*N+idy] - u[idx*N+idy+1] )/(h*h) - f[idx*N+idy];
}


int main(){
  long N = 6402; // dimension of 2D space
  double h = 1.0/(N+1); // size of update step
  double *u, *f, *err; // u is the solution, f is the forcing function
  hipHostMalloc((void**)&u, (N)*(N)*sizeof(double), hipHostMallocDefault);
  hipHostMalloc((void**)&f, (N)*(N)*sizeof(double), hipHostMallocDefault);
  hipHostMalloc((void**)&err, N*N*sizeof(double), hipHostMallocDefault);  

  for(long i = 0; i < N; i++){
    for(long j =0; j < N; j++) {
       u[i*N+j] = 0;
       f[i*N+j] = 1;
       //temp[i*N +j] = 0;
       err[i*N+j] = 0;
   }
  }

  dim3 blockDim(BLOCK_DIM-2,BLOCK_DIM-2);
  dim3 gridDim( (N-2)/(BLOCK_DIM-2),(N-2)/(BLOCK_DIM-2));
  double *u_d, *f_d, *temp_d, *err_d;
  hipMalloc(&u_d, N*N*sizeof(double));
  hipMalloc(&f_d, N*N*sizeof(double));
  hipMalloc(&temp_d, N*N*sizeof(double));
  hipMalloc(&err_d, N*N*sizeof(double));

  hipMemcpyAsync(u_d, u, N*N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(f_d, f, N*N*sizeof(double), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  long t = 1;
  double tt = omp_get_wtime();
  while (t < 10000) {
   jacobi_kernel_nsmem<<<gridDim, blockDim, 0>>>(u_d,f_d,err_d, temp_d, N, h );
  if ((t % 1000) == 0) { 
    printf("time per 1000 iterations = %f s \n ", (omp_get_wtime()-tt) );
    //printf("Bandwidth = %f\n", 1000*7*(N-2)*(N-2)/(omp_get_wtime()-tt)/1e9);
    hipMemcpyAsync(err,err_d,N*N*sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    printf("Err = %f\n", norm(err, N));
    tt = omp_get_wtime();
  }
  t+=1;
}
 hipFree(u_d);
 hipFree(f_d);
 hipFree(temp_d);
 hipFree(err_d);

 hipHostFree(u);
 hipHostFree(f);
 hipHostFree(err);

}

